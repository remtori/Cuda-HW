
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
			exit(EXIT_FAILURE);                                    \
		}                                                          \
	}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

void readPnm(char *fileName,
			 int &width, int &height, uchar3 *&pixels)
{
	FILE *f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);

	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);

	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height,
			  char *fileName)
{
	FILE *f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "P3\n%i\n%i\n255\n", width, height);

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);

	fclose(f);
}

__global__ void blurImgKernel(uchar3 *inPixels, int width, int height,
							  float *filter, int filterWidth,
							  uchar3 *outPixels)
{
	// TODO	
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	float3 sum = make_float3(0, 0, 0);
	for (int i = 0; i < filterWidth; i++) 
	{
		for (int j = 0; j < filterWidth; j++) 
		{
			int aX = x + (i - filterWidth / 2);
			int aY = y + (j - filterWidth / 2);
			if (aX < 0) aX = 0;
			if (aX >= width) aX = width - 1;
			if (aY < 0) aY = 0;
			if (aY >= height) aY = height - 1;

			sum.x += inPixels[aX + aY * width].x * filter[i];
			sum.y += inPixels[aX + aY * width].y * filter[i];
			sum.z += inPixels[aX + aY * width].z * filter[i];
		}
	}

	outPixels[x + y * width].x = sum.x;
	outPixels[x + y * width].y = sum.y;
	outPixels[x + y * width].z = sum.z;
}

void blurImg(uchar3 *inPixels, int width, int height, float *filter, int filterWidth,
			 uchar3 *outPixels,
			 bool useDevice = false, dim3 blockSize = dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
		for (int x = 0; x < width; x++)
		{
			for (int y = 0; y < height; y++)
			{
				float3 sum = make_float3(0, 0, 0);
				for (int i = 0; i < filterWidth; i++) 
				{
					for (int j = 0; j < filterWidth; j++) 
					{
						int aX = x + (i - filterWidth / 2);
						int aY = y + (j - filterWidth / 2);
						if (aX < 0) aX = 0;
						if (aX >= width) aX = width - 1;
						if (aY < 0) aY = 0;
						if (aY >= height) aY = height - 1;

						sum.x += inPixels[aX + aY * width].x * filter[i];
						sum.y += inPixels[aX + aY * width].y * filter[i];
						sum.z += inPixels[aX + aY * width].z * filter[i];
					}
				}

				outPixels[x + y * width].x = sum.x;
				outPixels[x + y * width].y = sum.y;
				outPixels[x + y * width].z = sum.z;
			}
		}
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
		float *d_filter;
		uchar3 *d_in, *d_out;
		CHECK(hipMalloc(&d_in, width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_out, width * height * sizeof(uchar3)));
		CHECK(hipMalloc(&d_filter, filterWidth * sizeof(float)));

		// TODO: Copy data to device memories
		CHECK(hipMemcpy(d_in, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_filter, filter, filterWidth * sizeof(float), hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		blurImgKernel<<<gridSize, blockSize>>>(d_in, width, height, d_filter, filterWidth, d_out);

		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_out, width * height * sizeof(uchar3), hipMemcpyDeviceToHost));

		// TODO: Free device memories
		CHECK(hipFree(d_in));
		CHECK(hipFree(d_out));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n",
		   useDevice == true ? "use device" : "use host", time);
}

float computeError(uchar3 *a1, uchar3 *a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char *concatStr(const char *s1, const char *s2)
{
	char *result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

int main(int argc, char **argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 *inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 *correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect
	int filterWidth = 9;
	float *filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 *hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);

	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 *deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char *outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
